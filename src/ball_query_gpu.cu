#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// 
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: query_xyz(b, m, 3) db_xyz(b, n, 3)
// output: idx(b, m, num_neighbors)
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int num_neighbors,
                                        const float *__restrict__ query_xyz,
                                        const float *__restrict__ db_xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  db_xyz += batch_index * n * 3;
  query_xyz += batch_index * m * 3;
  idx += m * num_neighbors * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = query_xyz[j * 3 + 0];
    float new_y = query_xyz[j * 3 + 1];
    float new_z = query_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < num_neighbors; ++k) {
      float x = db_xyz[k * 3 + 0];
      float y = db_xyz[k * 3 + 1];
      float z = db_xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < num_neighbors; ++l) {
            idx[j * num_neighbors + l] = k;
          }
        }
        idx[j * num_neighbors + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_wrapper(int b, int n, int m, float radius,
                                     int num_neighbors, const float *query_xyz,
                                     const float *db_xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, num_neighbors, query_xyz, db_xyz, idx);

  CUDA_CHECK_ERRORS();
}
